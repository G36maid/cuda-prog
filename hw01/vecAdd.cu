#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>

#define N (1 << 24) // 約 16M 筆資料
#define THREADS_PER_BLOCK 256

void initData(float *a, float *b, int n) {
    for (int i = 0; i < n; ++i) {
        a[i] = static_cast<float>(rand()) / RAND_MAX;
        b[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

void checkResult(const float *a, const float *b, int n) {
    const float epsilon = 1e-5;
    for (int i = 0; i < n; ++i) {
        if (fabs(a[i] - b[i]) > epsilon) {
            fprintf(stderr, "Mismatch at index %d: CPU=%.6f, GPU=%.6f\n", i, a[i], b[i]);
            return;
        }
    }
    printf("Result verification passed!\n");
}

void addVectorsCPU(const float *a, const float *b, float *c, int n) {
    for (int i = 0; i < n; ++i) {
        c[i] = a[i] + b[i];
    }
}

__global__ void addKernel(const float *a, const float *b, float *c, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void addVectorsGPU(const float *a, const float *b, float *c, int n) {
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    size_t size = n * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    addKernel<<<blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    float *a = (float *)malloc(N * sizeof(float));
    float *b = (float *)malloc(N * sizeof(float));
    float *c_cpu = (float *)malloc(N * sizeof(float));
    float *c_gpu = (float *)malloc(N * sizeof(float));

    initData(a, b, N);

    // CPU 計算
    auto start_cpu = std::chrono::high_resolution_clock::now();
    addVectorsCPU(a, b, c_cpu, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    double time_cpu = std::chrono::duration<double>(end_cpu - start_cpu).count();
    printf("CPU Time: %.4f seconds\n", time_cpu);

    // GPU 計算
    auto start_gpu = std::chrono::high_resolution_clock::now();
    addVectorsGPU(a, b, c_gpu, N);
    auto end_gpu = std::chrono::high_resolution_clock::now();
    double time_gpu = std::chrono::duration<double>(end_gpu - start_gpu).count();
    printf("GPU Time: %.4f seconds\n", time_gpu);

    // 驗證
    checkResult(c_cpu, c_gpu, N);

    free(a);
    free(b);
    free(c_cpu);
    free(c_gpu);
    return 0;
}
